
#include "CuEvent.hpp"
#include <hip/hip_runtime.h>

using namespace rnn;
using namespace rnn::cuda;

struct CuEvent::CuEventImpl {
  hipEvent_t event;

  CuEventImpl() {
    hipEventCreateWithFlags(&event, hipEventDisableTiming);
  }

  void *GetCudaEvent(void) {
    return &event;
  }
};

CuEvent::CuEvent() : impl(new CuEventImpl()) {}
CuEvent::~CuEvent() = default;

void *CuEvent::GetCudaEvent(void) {
  return impl->GetCudaEvent();
}
